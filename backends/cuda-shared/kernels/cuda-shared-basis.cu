#include <ceed/ceed.h>
#include <hip/hip_runtime.h>

const int sizeMax = 16;
__constant__ CeedScalar c_B[sizeMax*sizeMax];
__constant__ CeedScalar c_G[sizeMax*sizeMax];

//------------------------------------------------------------------------------
// Interp device initalization
//------------------------------------------------------------------------------
extern "C" int CeedCudaInitInterp(CeedScalar *d_B, CeedInt P1d, CeedInt Q1d,
                                  CeedScalar **c_B_ptr) {
  const int Bsize = P1d*Q1d*sizeof(CeedScalar);
  hipMemcpyToSymbol(HIP_SYMBOL(c_B), d_B, Bsize, 0, hipMemcpyDeviceToDevice);
  hipGetSymbolAddress((void **)c_B_ptr, c_B);

  return 0;
}

//------------------------------------------------------------------------------
// Grad device initalization
//------------------------------------------------------------------------------
extern "C" int CeedCudaInitInterpGrad(CeedScalar *d_B, CeedScalar *d_G,
    CeedInt P1d, CeedInt Q1d, CeedScalar **c_B_ptr, CeedScalar **c_G_ptr) {
  const int Bsize = P1d*Q1d*sizeof(CeedScalar);
  hipMemcpyToSymbol(HIP_SYMBOL(c_B), d_B, Bsize, 0, hipMemcpyDeviceToDevice);
  hipGetSymbolAddress((void **)c_B_ptr, c_B);
  hipMemcpyToSymbol(HIP_SYMBOL(c_G), d_G, Bsize, 0, hipMemcpyDeviceToDevice);
  hipGetSymbolAddress((void **)c_G_ptr, c_G);

  return 0;
}
//------------------------------------------------------------------------------
